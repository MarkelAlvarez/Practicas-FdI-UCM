#include "hip/hip_runtime.h"
#include <stdio.h>
#include "matrix_mul.h"

// Thread block size
#define BLOCK_SIZE 4 

// Forward declaration of the device multiplication function
__global__ void Muld(float*, float*, int, int, float*);

// Host multiplication function
// Compute C = A * B
// hA is the height of A
// wA is the width of A
// wB is the width of B

//export void Mul(float*, float*, int, int, int, float*);
void Mul(float* A, float* B, int hA, int wA, int wB, float* C)
{
	int size;

	// Load A and B to the device
	float* Ad;
	size = hA * wA * sizeof(float);
	hipMalloc((void**)&Ad, size);
	hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);

	float* Bd;
	size = wA * wB * sizeof(float);
	hipMalloc((void**)&Bd, size);
	hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

	// Allocate C on the device
	float* Cd;
	size = hA * wB * sizeof(float);
	hipMalloc((void**)&Cd, size);

	// Compute the execution configuration assuming
	// the matrix dimensions are multiples of BLOCK_SIZE
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(wB / dimBlock.x, hA / dimBlock.y);

	// Launch the device computation
	Muld<<<dimGrid, dimBlock>>>(Ad, Bd, wA, wB, Cd);

	// Read C from the device
	hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
}

// Device multiplication function called by Mul()
// Compute C = A * B
// wA is the width of A
// wB is the width of B
__global__ void Muld(float* A, float* B, int wA, int wB, float* C)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first sub-matrix of A processed by the block
	int aBegin = BLOCK_SIZE * wA * by;

	// Index of the last sub-matrix of A processed by the block
	int aEnd = aBegin + wA - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the sub-matrices of B
	int bStep = BLOCK_SIZE * wB;

	// The element of the block sub-matrix that is computed
	// by the thread
	float Csub = 0;

	// Loop over all the sub-matrices of A and B required to
	// compute the block sub-matrix
	for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
	{
		// Shared memory for the sub-matrix of A
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

		// Shared memory for the sub-matrix of B
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from global memory to shared memory;
		// each thread loads one element of each matrix
		As[ty][tx] = A[a + wA * ty + tx];
		Bs[ty][tx] = B[b + wB * ty + tx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
		for (int k = 0; k < BLOCK_SIZE; ++k)
		{
			Csub += As[ty][k] * Bs[k][tx];
		}

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}
	
	// Write the block sub-matrix to global memory;
	// each thread writes one element
	int bOut = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	C[bOut + wB * ty + tx] = Csub;
}