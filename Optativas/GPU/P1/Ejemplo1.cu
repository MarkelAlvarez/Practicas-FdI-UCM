#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
//CUDA
#include <hip/hip_runtime.h>

double wtime(void)
{
	static struct timeval tv0;
	double time_;

	gettimeofday(&tv0,(struct timezone*)0);
	time_ = (double)((tv0.tv_usec + (tv0.tv_sec)*1000000));
	return (time_ / 1000000);
}

void addMatrix(float *a, float *b, float *c, int N)
{
	int i, j, idx;

	for (i=0; i<N; i++)
	{
		for(j=0; j<N;j++)
		{
			idx = i*N+j;
			a[idx]=b[idx]+c[idx];
		}
	}
} 

__global__ void addMatrixGPU(float *a, float *b, float *c, int N )
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int idx;

	if (i < N && j < N)
	{
		idx = i*N+j;
		a[idx]=b[idx]+c[idx];
	}
}

int main(int argc, char *argv[])
{
	int i, j, N;
	double t0, t1;
	float *a, *b, *c, *a_host;
	float *a_GPU, *b_GPU, *c_GPU;

	if(argc>1)
	{
		N = atoi(argv[1]); printf("N=%i\n", N);
	}
	else
	{
		printf("Error!!!! \n ./exec number\n");
		return (0);
	}

	// Mallocs CPU
	a  = (float *)malloc(sizeof(float)*N*N);
	b  = (float *)malloc(sizeof(float)*N*N);
	c  = (float *)malloc(sizeof(float)*N*N);

	for (i=0; i<N*N; i++)
	{
		b[i] = i-1;
		c[i] = i;
	}

	/*****************/
	/* Add Matrix CPU*/
	/*****************/
	t0 = wtime();
	addMatrix(a, b, c, N);
	t1 = wtime();
	printf("Time CPU=%f\n", t1-t0);

	/* Mallocs GPU */
	hipMalloc((void **)&a_GPU, sizeof(float)*N*N);
	hipMalloc((void **)&b_GPU, sizeof(float)*N*N);
	hipMalloc((void **)&c_GPU, sizeof(float)*N*N);

	/* CPU->GPU */
	hipMemcpy(b_GPU, b, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(c_GPU, c, sizeof(float)*N*N, hipMemcpyHostToDevice);

	/*****************/
	/* Add Matrix GPU*/
	/*****************/
	dim3 dimBlock(16, 16, 1);
	dim3 dimGrid(ceil(N/16), ceil(N/16), 1);
	t0 = wtime();
	addMatrixGPU<<<dimGrid,dimBlock>>>(a_GPU, b_GPU, c_GPU, N);
	hipDeviceSynchronize();
	t1 = wtime(); printf("Time GPU=%f\n", t1-t0);

	/* GPU->CPU */
	a_host  = (float *)malloc(sizeof(float)*N*N);
	hipMemcpy(a_host, a_GPU, sizeof(float)*N*N, hipMemcpyDeviceToHost);

	/************/
	/* Results  */
	/************/
	for (i=0; i<N; i++)
	{
		for (j=0; j<N; j++)
		{
			if(fabs(a[i*N+j]-a_host[i*N+j])>1e-5)
			{
				printf("a!=a_host in (%i,%i): ", i,j);
				printf("A[%i][%i] = %f A_GPU[%i][%i]=%f\n", i, j, a[i*N+j], i, j, a_host[i*N+j]);
			}
		}	
	}

	/* Free CPU */
	free(a);
	free(b);
	free(c);
	free(a_host);

	/* Free GPU */
	hipFree(a_GPU);
	hipFree(b_GPU);
	hipFree(c_GPU);

	return(1);
}