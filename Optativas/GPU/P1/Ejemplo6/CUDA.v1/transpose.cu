#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

/* Time */
#include <sys/time.h>
#include <sys/resource.h>

static struct timeval tv0;

double getMicroSeconds()
{
	double t;
	gettimeofday(&tv0, (struct timezone*)0);
	t = ((tv0.tv_usec) + (tv0.tv_sec)*1000000);

	return (t);
}

void init_seed()
{
	FILE *fd;
	int seedi=1;

	/* Generated random values between 0.00 - 1.00 */
	fd = fopen("/dev/urandom", "r");
	fread(&seedi, sizeof(int), 1, fd);
	fclose(fd);
	srand(seedi);
}

void init1Drand(float *buffer, int n)
{
	int i, j;

	for (i=0; i<n; i++)
	{
		for(j=0; j<n; j++)
		{
			buffer[i*n+j] = 500.0*(float(rand())/RAND_MAX)-500.0; /* [-500 500]*/
		}
	}
}

float *getmemory1D( int nx )
{
	int i;
	float *buffer;

	if((buffer=(float *)malloc(nx*sizeof(float *)))== NULL)
	{
		fprintf(stderr, "ERROR in memory allocation\n");

		return(NULL);
	}

	for(i=0; i<nx; i++)
	{
		buffer[i] = 0.0;
	}

	return(buffer);
}

/*
 * Traspose 1D version
 */
void transpose1D(float *in, float *out, int n)
{
	int i, j;

	for(j=0; j < n; j++)
	{
		for(i=0; i < n; i++)
		{
			out[j*n+i] = in[i*n+j]; 
		}
	}
}

/*
 * Traspose CUDA version
 */
#define NTHREADS1D 256

__global__ void transpose_device(float *in, float *out, int rows, int cols) 
{ 
	int i, j; 
	i = blockIdx.x * blockDim.x + threadIdx.x; 

	if (i<rows)
	{
		for ( j=0; j<cols; j++) 
		{
			out [ i * rows + j ] = in [ j * cols + i ]; 
		}
	}
		
}

int check(float *GPU, float *CPU, int n)
{
	int i;

	for (i=0; i<n; i++)
	{
		if(GPU[i]!=CPU[i])
		{
			printf("(%i) %f %f\n", i, GPU[i], CPU[i]);

			return(1);
		}
	}
		
	return(0);
}

int main(int argc, char **argv)
{
	int n;
	double t0, t1;
	float *array1D_trans;
	float *darray1D, *darray1D_trans;

	if (argc==2)
	{
		n = atoi(argv[1]);
	}
	else
	{
		n = 8192;
		printf("./exec n (by default n=%i)\n", n);
	}
	
	/* Initizalization */
	init_seed();

	/* Transpose 1D version */
	hipMallocManaged((void**)&darray1D, n*n*sizeof(float));
	array1D_trans = (float*)malloc(sizeof(float)*n*n);
	t0 = getMicroSeconds();
	transpose1D(darray1D, array1D_trans, n);
	printf("Transpose version 1D: %f MB/s\n", n*n*sizeof(float)/((getMicroSeconds()-t0)/1000000)/1024/1024);

	/* CUDA vesion */
	hipMallocManaged((void**)&darray1D_trans, n*n*sizeof(float));

	dim3 dimBlock(NTHREADS1D);
	int blocks = n/NTHREADS1D;
	if (n%NTHREADS1D>0)
	{
		blocks++;
	}
	dim3 dimGrid(blocks);

	t0 = getMicroSeconds();
	transpose_device<<<dimGrid,dimBlock>>>(darray1D, darray1D_trans, n, n);	
	hipDeviceSynchronize();
	t1 = getMicroSeconds();

	printf("Transpose kernel version: %f MB/s tKernel=%f (us)\n", n*n*sizeof(float)/((getMicroSeconds()-t0)/1000000)/1024/1024, (t1-t0)/1000000);
	
	if (check(darray1D_trans, array1D_trans, n*n))
	{
		printf("Transpose CPU-GPU differs!!\n");
	}

	hipFree(darray1D);
	hipFree(darray1D_trans);

	return(0);
}