#include <stdio.h>
#include "hipblas.h"
#include "matrix_mul.h"

// Host multiplication function
// Compute C = A * B
// hA is the height of A
// wA is the width of A
// wB is the width of B

extern "C"
void Mul(float* A, float* B, int hA, int wA, int wB, float* C)
{
	int size;

	// Load A and B to the device
	float* Ad;
	size = hA * wA * sizeof(float);
	hipMalloc((void**)&Ad, size);
	hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);

	float* Bd;
	size = wA * wB * sizeof(float);
	hipMalloc((void**)&Bd, size);
	hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

	// Allocate C on the device
	float* Cd;
	size = hA * wB * sizeof(float);
	hipMalloc((void**)&Cd, size);

	// Compute the execution configuration
	hipblasSgemm( 'N', 'N',
		hA,			/* [m] */ 
		wB,			/* [n] */  
		wA,			/* [k] */ 
		1,			/* alfa */ 
		Ad, hA,		/* A[m][k], num columnas (lda) */ 
		Bd, wA,		/* B[k][n], num columnas (ldb) */
		0,			/* beta */
		Cd, hA		/* C[m][n], num columnas (ldc) */
	);

	// Read C from the device
	hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
}