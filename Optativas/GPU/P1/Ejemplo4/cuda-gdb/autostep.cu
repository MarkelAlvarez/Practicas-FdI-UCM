#include "hip/hip_runtime.h"
#define NUM_BLOCKS 8
#define THREADS_PER_BLOCK 64

__global__ void example(int **data) {
    int value1, value2, value3, value4, value5;
    int idx1, idx2, idx3;

    idx1 = blockIdx.x * blockDim.x;
    idx2 = threadIdx.x;
    idx3 = idx1 + idx2;
    value1 = *(data[idx1]);
    value2 = *(data[idx2]);
    value3 = value1 + value2;
    value4 = value1 * value2;
    value5 = value3 + value4;
    *(data[idx3]) = value5;
    *(data[idx1]) = value3;
    *(data[idx2]) = value4;
    idx1 = idx2 = idx3 = 0;
}

int main(int argc, char *argv[]) {
    int *host_data[NUM_BLOCKS*THREADS_PER_BLOCK];
    int **dev_data;
    const int zero = 0;

    /* Allocate an integer for each thread in each block */
    for (int block = 0; block < NUM_BLOCKS; block++) {
        for (int thread = 0; thread < THREADS_PER_BLOCK; thread++) {
            int idx = thread + block * THREADS_PER_BLOCK;
            hipMalloc(&host_data[idx], sizeof(int));
            hipMemcpy(host_data[idx], &zero, sizeof(int), hipMemcpyHostToDevice);
        }
    }

    /* This inserts an error into block 3, thread 33*/
    host_data[3*THREADS_PER_BLOCK + 33] = NULL;

    /* Copy the array of pointers to the device */
    hipMalloc((void**)&dev_data, sizeof(host_data));
    hipMemcpy(dev_data, host_data, sizeof(host_data), hipMemcpyHostToDevice);

    /* Execute example */
    example <<< NUM_BLOCKS, THREADS_PER_BLOCK >>> (dev_data);
    hipDeviceSynchronize();
}

