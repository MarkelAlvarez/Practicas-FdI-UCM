#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "routinesGPU.h"
#define BLOCK_SIZE 16
#define DEG2RAD 0.017453f

void getlines(int threshold, uint32_t *accumulators, int accu_width, int accu_height, int width, int height, float *sin_table, float *cos_table, int *x1_lines, int *y1_lines, int *x2_lines, int *y2_lines, int *lines)
{
	int rho, theta, ii, jj;
	uint32_t max;

	for(rho = 0; rho < accu_height; rho++)
	{
		for(theta = 0; theta < accu_width; theta++)  
		{  
			if(accumulators[(rho*accu_width) + theta] >= threshold)  
			{  
				//Is this point a local maxima (9x9)  
				max = accumulators[(rho*accu_width) + theta]; 
				for(int ii = -4; ii <= 4; ii++)  
				{  
					for(int jj = -4; jj <= 4; jj++)  
					{  
						if( (ii + rho >= 0 && ii + rho < accu_height) && (jj + theta >= 0 && jj + theta < accu_width))  
						{  
							if(accumulators[((rho + ii) * accu_width) + (theta + jj)] > max)  
							{
								max = accumulators[((rho + ii) * accu_width) + (theta + jj)];
							}  
						}  
					}  
				}  

				if(max == accumulators[(rho * accu_width) + theta]) //local maxima
				{
					int x1, y1, x2, y2;  
					x1 = y1 = x2 = y2 = 0;  

					if(theta >= 45 && theta <= 135)  
					{
						if (theta>90)
						{
							//y = (r - x cos(t)) / sin(t)  
							x1 = width/2;  
							y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2 = width;  
							y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);  
						}
						else
						{
							//y = (r - x cos(t)) / sin(t)  
							x1 = 0;  
							y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2 = width*2/5;  
							y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2); 
						}
					}
					else
					{
						//x = (r - y sin(t)) / cos(t);  
						y1 = 0;  
						x1 = ((float)(rho-(accu_height/2)) - ((y1 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
						y2 = height;  
						x2 = ((float)(rho-(accu_height/2)) - ((y2 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
					}
					
					x1_lines[*lines] = x1;
					y1_lines[*lines] = y1;
					x2_lines[*lines] = x2;
					y2_lines[*lines] = y2;
					(*lines)++;
				}
			}
		}
	}
}

void init_cos_sin_table(float *sin_table, float *cos_table, int n)
{
	int i;

	for (i=0; i<n; i++)
	{
		sin_table[i] = sinf(i*DEG2RAD);
		cos_table[i] = cosf(i*DEG2RAD);
	}
}

__global__ void calculateNR(uint8_t *im, float *NR, int height, int width)
{
	int i = blockIdx.y*blockDim.y + threadIdx.y+2;
	int j = blockIdx.x*blockDim.x + threadIdx.x+2;
	int col = threadIdx.x+2;
	int row = threadIdx.y+2;	
	__shared__ uint8_t imgCompartida[BLOCK_SIZE + 4][BLOCK_SIZE + 4];
	
	imgCompartida[threadIdx.y + 2][threadIdx.x + 2] = im[(i)*width+(j)];
	
	if(!threadIdx.x)
	{
		imgCompartida[threadIdx.y+2][threadIdx.x+1] = im[(i)*width+(j-1)];
		imgCompartida[threadIdx.y+2][threadIdx.x] = im[(i)*width+(j-2)];
	}
	if(!threadIdx.y)
	{
		imgCompartida[threadIdx.y+1][threadIdx.x+2] = im[(i-1)*width+(j)];
		imgCompartida[threadIdx.y][threadIdx.x+2] = im[(i-2)*width+(j)];
	}
	if(threadIdx.x == BLOCK_SIZE-1)
	{
		imgCompartida[threadIdx.y+2][threadIdx.x+3] = im[(i)*width+(j+1)];
		imgCompartida[threadIdx.y+2][threadIdx.x+4] = im[(i)*width+(j+2)];
	}
	if(threadIdx.y == BLOCK_SIZE-1)
	{
		imgCompartida[threadIdx.y+3][threadIdx.x+2] = im[(i+1)*width+(j)];
		imgCompartida[threadIdx.y+4][threadIdx.x+2] = im[(i+2)*width+(j)];
	}
	
	if(((i >= 2) && (i < height - 2)) && ((j >= 2) && (j < width-2)))
	{
		NR[i*width+j] =
				 (2.0*imgCompartida[row-2][col-2] 	+  4.0*imgCompartida[row-2][col-1] 	+  5.0*imgCompartida[row-2][col] 	+  4.0*imgCompartida[row-2][col+1] 	+ 2.0*imgCompartida[row-2][col+2]
				+ 4.0*imgCompartida[row-1][col-2] 	+  9.0*imgCompartida[row-1][col-1] 	+ 12.0*imgCompartida[row-1][col] 	+  9.0*imgCompartida[row-1][col+1] 	+ 4.0*imgCompartida[row-1][col+2]
				+ 5.0*imgCompartida[row][col-2] 	+ 12.0*imgCompartida[row][col-1] 	+ 15.0*imgCompartida[row][col] 		+ 12.0*imgCompartida[row][col+1] 	+ 5.0*imgCompartida[row][col-2]
				+ 4.0*imgCompartida[row+1][col-2] 	+  9.0*imgCompartida[row+1][col-1] 	+ 12.0*imgCompartida[row+1][col] 	+  9.0*imgCompartida[row+1][col+1] 	+ 4.0*imgCompartida[row+1][col+2]
				+ 2.0*imgCompartida[row+2][col-2] 	+  4.0*imgCompartida[row+2][col-1] 	+  5.0*imgCompartida[row+2][col] 	+  4.0*imgCompartida[row+2][col+1]	+ 2.0*imgCompartida[row+2][col+2])
				/159.0;

	}
}

__global__ void calculateGPhi(float *NR, float *G, float *phi, float *Gx, float *Gy, int height, int width)
{
	int i, j;
	float PI = 3.141593;
		
	i = blockIdx.y * blockDim.y + threadIdx.y;
	j = blockIdx.x * blockDim.x + threadIdx.x;

	phi[i*width+j] = 0;

	if(((i >=2) && (i < height-2)) && ((j >=2) && (j < width-2))) {
		// Intensity gradient of the image
		Gx[i*width+j] = 
					(1.0*NR[(i-2)*width+(j-2)] 	+  2.0*NR[(i-2)*width+(j-1)] +  (-2.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
					+ 4.0*NR[(i-1)*width+(j-2)] +  8.0*NR[(i-1)*width+(j-1)] +  (-8.0)*NR[(i-1)*width+(j+1)] + (-4.0)*NR[(i-1)*width+(j+2)]
					+ 6.0*NR[(i  )*width+(j-2)] + 12.0*NR[(i  )*width+(j-1)] + (-12.0)*NR[(i  )*width+(j+1)] + (-6.0)*NR[(i  )*width+(j+2)]
					+ 4.0*NR[(i+1)*width+(j-2)] +  8.0*NR[(i+1)*width+(j-1)] +  (-8.0)*NR[(i+1)*width+(j+1)] + (-4.0)*NR[(i+1)*width+(j+2)]
					+ 1.0*NR[(i+2)*width+(j-2)] +  2.0*NR[(i+2)*width+(j-1)] +  (-2.0)*NR[(i+2)*width+(j+1)] + (-1.0)*NR[(i+2)*width+(j+2)]);


		Gy[i*width+j] = 
					((-1.0)*NR[(i-2)*width+(j-2)] 	+ (-4.0)*NR[(i-2)*width+(j-1)]	+ (-6.0)*NR[(i-2)*width+(j)] 	+ (-4.0)*NR[(i-2)*width+(j+1)]	+ (-1.0)*NR[(i-2)*width+(j+2)]
					+ (-2.0)*NR[(i-1)*width+(j-2)] 	+ (-8.0)*NR[(i-1)*width+(j-1)]	+ (-12.0)*NR[(i-1)*width+(j)] 	+ (-8.0)*NR[(i-1)*width+(j+1)] 	+ (-2.0)*NR[(i-1)*width+(j+2)]
					+ 2.0*NR[(i+1)*width+(j-2)] 	+ 8.0*NR[(i+1)*width+(j-1)] 	+ 12.0*NR[(i+1)*width+(j)] 		+ 8.0*NR[(i+1)*width+(j+1)] 	+ 2.0*NR[(i+1)*width+(j+2)]
					+ 1.0*NR[(i+2)*width+(j-2)] 	+ 4.0*NR[(i+2)*width+(j-1)] 	+ 6.0*NR[(i+2)*width+(j)] 		+ 4.0*NR[(i+2)*width+(j+1)] 	+ 1.0*NR[(i+2)*width+(j+2)]);

		G[i*width+j]   = sqrtf((Gx[i*width+j]*Gx[i*width+j])+(Gy[i*width+j]*Gy[i*width+j]));	//G = √Gx²+Gy²
		phi[i*width+j] = atan2f(fabs(Gy[i*width+j]), fabs(Gx[i*width+j]));

		if(fabs(phi[i*width+j])<=PI/8)
		{
			phi[i*width+j] = 0;
		}
			
		else if (fabs(phi[i*width+j])<= 3*(PI/8))
		{
			phi[i*width+j] = 45;
		}
			
		else if (fabs(phi[i*width+j]) <= 5*(PI/8))
		{
			phi[i*width+j] = 90;
		}
			
		else if (fabs(phi[i*width+j]) <= 7*(PI/8))
		{
			phi[i*width+j] = 135;
		}
		else
		{
			phi[i*width+j] = 0;
		}	
	}
}

__global__ void calculatePedge(float *G, float *phi, /*uint8_t*/ int *pedge, int height, int width)
{
	int i, j;

	i = blockIdx.y * blockDim.y + threadIdx.y;
	j = blockIdx.x * blockDim.x + threadIdx.x;
	
	pedge[i*width+j] = 0;

	if(((i >=3) && (i < height-3)) && ((j >=3) && (j < width-3)))
	{
		if(phi[i*width+j] == 0)
		{
			if(G[i*width+j]>G[i*width+j+1] && G[i*width+j]>G[i*width+j-1]) //edge is in N-S
			{
				pedge[i*width+j] = 1;
			}

		}
		else if(phi[i*width+j] == 45)
		{
			if(G[i*width+j]>G[(i+1)*width+j+1] && G[i*width+j]>G[(i-1)*width+j-1]) // edge is in NW-SE
			{
				pedge[i*width+j] = 1;
			}

		}
		else if(phi[i*width+j] == 90)
		{
			if(G[i*width+j]>G[(i+1)*width+j] && G[i*width+j]>G[(i-1)*width+j]) //edge is in E-W
			{
				pedge[i*width+j] = 1;
			}

		}
		else if(phi[i*width+j] == 135)
		{
			if(G[i*width+j]>G[(i+1)*width+j-1] && G[i*width+j]>G[(i-1)*width+j+1]) // edge is in NE-SW
			{
				pedge[i*width+j] = 1;
			}
		}
	}
}

__global__ void calculateImageOut(uint8_t *image_out, float *G, int *pedge, float level, int height, int width)
{
	int i, j;
	int ii, jj;
	float lowthres = level/2;
	float hithres = 2*(level);
	
	i = blockIdx.y * blockDim.y + threadIdx.y;
	j = blockIdx.x * blockDim.x + threadIdx.x;

	// Hysteresis Thresholding
	image_out[i*width+j] = 0;

	if(((i >=3) && (i < height-3)) && ((j >=3) && (j < width-3)))
	{
		if(G[i*width+j]>hithres && pedge[i*width+j])
		{
			image_out[i*width+j] = 255;
		}
		else if(pedge[i*width+j] && G[i*width+j]>=lowthres && G[i*width+j]<hithres)
		{
			// check neighbours 3x3
			for (ii=-1;ii<=1; ii++)
			{
				for (jj=-1;jj<=1; jj++)
				{
					if (G[(i+ii)*width+j+jj]>hithres)
					{
						image_out[i*width+j] = 255;
					}
				}
			}
		}
	}
}

void cannyGPU(uint8_t *im, uint8_t *image_out, float*NR, float *G, float *phi, float *Gx, float *Gy, /*uint8_t*/ int *pedge, float level, int height, int width)
{
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	
	int hBlocks = height/BLOCK_SIZE;
	if(height % BLOCK_SIZE != 0)
	{
		hBlocks++;
	}

	int wBlocks = width / BLOCK_SIZE;
	if (width % BLOCK_SIZE!= 0)
	{
		wBlocks + 1;
	}

	dim3 blocks(hBlocks, wBlocks);
	
	calculateNR<<<blocks, threads>>>(im, NR, height, width);
	hipDeviceSynchronize();

	calculateGPhi<<<blocks,threads>>>(NR, G, phi, Gx, Gy, height, width);
	hipDeviceSynchronize();
	
	calculatePedge<<<blocks,threads>>>(G, phi, pedge, height, width);
	hipDeviceSynchronize();
	
	calculateImageOut<<<blocks,threads>>>(image_out, G, pedge, level, height, width);
}

void houghtransform(uint8_t *im, int width, int height, uint32_t *accumulators, int accu_width, int accu_height, float *sin_table, float *cos_table)
{
	int i, j, theta;

	float hough_h = ((sqrt(2.0) * (float)(height>width?height:width)) / 2.0);

	for(i=0; i<accu_width*accu_height; i++)
		accumulators[i]=0;	

	float center_x = width/2.0; 
	float center_y = height/2.0;

	for(i=0;i<height;i++)  
	{  
		for(j=0;j<width;j++)  
		{  
			if( im[ (i*width) + j] > 250 ) // Pixel is edge  
			{  
				for(theta=0;theta<180;theta++)  
				{  
					float rho = ( ((float)j - center_x) * cos_table[theta]) + (((float)i - center_y) * sin_table[theta]);
					accumulators[ (int)((round(rho + hough_h) * 180.0)) + theta]++;

				} 
			} 
		} 
	}
}

void line_asist_GPU(uint8_t *im, int height, int width, uint8_t *imEdge, float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge, float *sin_table, float *cos_table, uint32_t *accum, int accu_height, int accu_width, int *x1, int *x2, int *y1, int *y2, int *nlines)
{
	int threshold, size = height * width;
	
	uint8_t* matrix;
	hipMalloc((uint8_t**)&matrix, size * sizeof(uint8_t));
	hipMemcpy(matrix, im, size * sizeof(uint8_t), hipMemcpyHostToDevice);

	//reservamos memoria para la salida de canny y las demas matrices
	uint8_t* matrixOut;
	hipMalloc((uint8_t**)&matrixOut, size  * sizeof(uint8_t));

	float* noiseReduction_GPU;
	hipMalloc((float**)&noiseReduction_GPU, size  * sizeof(float));
	
	float* G_GPU;
	hipMalloc((float**)&G_GPU, size * sizeof(float));

	float* phi_GPU;
	hipMalloc((float**)&phi_GPU, size * sizeof(float));

	float* gx_GPU;
	hipMalloc((float**)&gx_GPU, size * sizeof(float));

	float* gy_GPU;
	hipMalloc((float**)&gy_GPU, size * sizeof(float));

	//uint8_t* pedge_GPU; 
	//hipMalloc((float**)&pedge_GPU, size * sizeof(uint8_t));
	int *pedge_GPU; 
	hipMalloc((float**)&pedge_GPU, size * sizeof(int));

	uint8_t* accum_GPU;
	hipMalloc(&accum_GPU, accu_width * accu_height * sizeof(uint32_t));

	cannyGPU(matrix, matrixOut, noiseReduction_GPU, G_GPU, phi_GPU, gx_GPU, gy_GPU, pedge_GPU, 1000.0f, height, width);

	/* hough transform */
	houghtransform(imEdge, width, height, accum, accu_width, accu_height, sin_table, cos_table);
	hipMemcpy(accum, accum_GPU, accu_width * accu_height * sizeof(uint32_t), hipMemcpyDeviceToHost);
	
	if (width > height)
	{
		threshold = width/6;
	}
	else
	{
		threshold = height/6;
	}

	hipMemcpy(matrix, matrixOut, size * sizeof(float),hipMemcpyDeviceToHost);

	getlines(threshold, accum, accu_width, accu_height, width, height, sin_table, cos_table, x1, y1, x2, y2, nlines);
}